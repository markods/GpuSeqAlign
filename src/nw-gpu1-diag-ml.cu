#include "hip/hip_runtime.h"
#include "common.hpp"


// cuda kernel for the parallel implementation
__global__ static void Nw_Gpu1_Kernel(
   const int* const seqX_gpu,
   const int* const seqY_gpu,
         int* const score_gpu,
   const int* const subst_gpu,
   const int adjrows,
   const int adjcols,
   const int substsz,
   const int indel,
   const int d   // the current minor diagonal in the score matrix (exclude the header row and column)
)
{
   // the dimensions of the matrix without its row and column header
   const int rows = -1 + adjrows;
   const int cols = -1 + adjcols;
   

   //  x x x x x x       x x x x x x       x x x x x x
   //  x / / / . .       x . . . / /       x . . . . .|/ /
   //  x / / . . .   +   x . . / / .   +   x . . . . /|/
   //  x / . . . .       x . / / . .       x . . . / /|

   // (d,p) -- element coordinates on the score matrix diagonal
   int pbeg = max( 0, d - (cols-1) );
   int pend = min( d, rows-1 );
   // position of the current thread's element on the matrix diagonal
   int p = pbeg + ( blockDim.x*blockIdx.x + threadIdx.x );


   // if the thread maps onto an element on the current matrix diagonal
   if( p <= pend )
   {
      // position of the current element
      int i = 1 + (   p );
      int j = 1 + ( d-p );


      // if the thread maps onto the start of the diagonal
      if( d < cols && p == 0 )
      {
         // initialize TOP header element
         el(score_gpu,adjcols, 0,j) = j*indel;
         // if this is also the zeroth diagonal (with only one element on it)
         if( d == 0 )
         {
            // initialize TOP-LEFT header element
            el(score_gpu,adjcols, 0,0) = 0*indel;
         }
      }
      // if the thread maps onto the end of the diagonal
      if( d < rows && p == pend )
      {
         // initialize LEFT header element
         el(score_gpu,adjcols, i,0) = i*indel;
      }

      // calculate the current element's value
      // +   always subtract the insert delete cost from the result, since that value was added to the initial temporary
      int p0 = el(subst_gpu,substsz, seqY_gpu[i], seqX_gpu[j]) - indel;
      
      int p1 =      el(score_gpu,adjcols, i-1,j-1) + p0;     // MOVE DOWN-RIGHT
      int p2 = max( el(score_gpu,adjcols, i-1,j  ) , p1 );   // MOVE DOWN
      int p3 = max( el(score_gpu,adjcols, i  ,j-1) , p2 );   // MOVE RIGHT
      el(score_gpu,adjcols, i,j) = p3 + indel;
   }
}



// parallel gpu implementation of the Needleman-Wunsch algorithm
NwStat NwAlign_Gpu1_Diag_Ml( NwParams& pr, NwInput& nw, NwResult& res )
{
   // number of threads per block
   // +   the tile is one-dimensional
   unsigned threadsPerBlock;

   // get the parameter values
   try
   {
      threadsPerBlock = pr["threadsPerBlock"].curr();
   }
   catch( const std::out_of_range& ex )
   {
      return NwStat::errorInvalidValue;
   }
 
   // adjusted gpu score matrix dimensions
   // +   the matrix dimensions are rounded up to 1 + the nearest multiple of the tile A size (in order to be evenly divisible)
   int adjrows = nw.adjrows;
   int adjcols = nw.adjcols;
   // special case when very small and very large sequences are compared
   if( adjrows == 1 ) { adjrows = 2; }
   if( adjcols == 1 ) { adjcols = 2; }
   // the dimensions of the matrix without its row and column header
   int rows = -1 + adjrows;
   int cols = -1 + adjcols;

   // start the timer
   res.sw.start();


   // reserve space in the ram and gpu global memory
   try
   {
      nw.seqX_gpu .init(         adjcols );
      nw.seqY_gpu .init( adjrows         );
      nw.score_gpu.init( adjrows*adjcols );

      nw.score    .init( nw.adjrows*nw.adjcols );
   }
   catch( const std::exception& ex )
   {
      return NwStat::errorMemoryAllocation;
   }

   // measure allocation time
   res.sw.lap( "alloc" );

   
   // copy data from host to device
   // +   gpu padding remains uninitialized, but this is not an issue since padding is only used to simplify kernel code (optimization)
   if( hipSuccess != ( cudaStatus = memTransfer( nw.seqX_gpu, nw.seqX, nw.adjcols ) ) )
   {
      return NwStat::errorMemoryTransfer;
   }
   if( hipSuccess != ( cudaStatus = memTransfer( nw.seqY_gpu, nw.seqY, nw.adjrows ) ) )
   {
      return NwStat::errorMemoryTransfer;
   }

   // measure memory transfer time
   res.sw.lap( "mem-to-device" );



   //  x x x x x x       x x x x x x       x x x x x x
   //  x / / / . .       x . . . / /       x . . . . .|/ /
   //  x / / . . .   +   x . . / / .   +   x . . . . /|/
   //  x / . . . .       x . / / . .       x . . . / /|
   // launch kernel for each minor diagonal of the score matrix
   {
      // grid and block dimensions for kernel
      dim3 gridA {};
      dim3 blockA {};

      // calculate size of shared memory per block in bytes
      int shmemsz = (
         0
      );

      // for all minor diagonals in the score matrix (excluding the header row and column)
      for( int d = 0;   d < cols-1 + rows;   d++ )
      {
         // calculate grid and block dimensions for kernel
         {
            int pbeg = max( 0, d - (cols-1) );
            int pend = min( d, rows-1 );
            
            // the number of elements on the current diagonal
            int dsize = pend-pbeg + 1;

            // take the number of threads per block as the only dimension
            blockA.x = threadsPerBlock;
            // take the number of blocks on the current score matrix diagonal as the only dimension
            // +   launch at least one block on the x axis
            gridA.x = ceil( float( dsize ) / threadsPerBlock );
         }


         // create variables for gpu arrays in order to be able to take their addresses
         int* seqX_gpu = nw.seqX_gpu.data();
         int* seqY_gpu = nw.seqY_gpu.data();
         int* score_gpu = nw.score_gpu.data();
         int* subst_gpu = nw.subst_gpu.data();
         
         // group arguments to be passed to kernel
         void* kargs[]
         {
            &seqX_gpu,
            &seqY_gpu,
            &score_gpu,
            &subst_gpu,
            &adjrows,
            &adjcols,
            &nw.substsz,
            &nw.indel,
            &d
         };
         
         // launch the kernel in the given stream (don't statically allocate shared memory)
         if( hipSuccess != ( cudaStatus = hipLaunchKernel( ( void* )Nw_Gpu1_Kernel, gridA, blockA, kargs, shmemsz, nullptr/*stream*/ ) ) )
         {
            return NwStat::errorKernelFailure;
         }
      }
   }

   // wait for the gpu to finish before going to the next step
   if( hipSuccess != ( cudaStatus = hipDeviceSynchronize() ) )
   {
      return NwStat::errorKernelFailure;
   }

   // measure calculation time
   res.sw.lap( "calc" );


   // save the calculated score matrix
   if( hipSuccess != ( cudaStatus = memTransfer( nw.score, nw.score_gpu, nw.adjrows, nw.adjcols, adjcols ) ) )
   {
      return NwStat::errorMemoryTransfer;
   }

   // measure memory transfer time
   res.sw.lap( "mem-to-host" );

   return NwStat::success;
}





