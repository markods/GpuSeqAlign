#include "hip/hip_runtime.h"
#include "defer.hpp"
#include "nw_fns.hpp"
#include "run_types.hpp"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <vector>

// Initialize the score matrix's header row and column.
// The score matrix is represented as two matrices (row-major order):
// + tile header row matrix,
// + tile header column matrix.
__global__ static void Nw_Gpu9_KernelA(
    int* const tileHrowMat_gpu,
    int* const tileHcolMat_gpu,
    const int trows,
    const int tcols,
    const int tileBx,
    const int tileBy,
    const int gapoCost)
{
    int tid = (blockDim.x * blockIdx.x + threadIdx.x);

    // Initialize score matrix header row.
    {
        // The tile header row and column have an extra zeroth element that needs initializing.
        // That's why we divide by (1 + ...).
        int jTile = tid / (1 + tileBx);
        int iTile = 0;

        if (jTile < tcols)
        {
            int jTileElem = tid % (1 + tileBx);
            int j = jTile * tileBx + jTileElem;

            int kHrow = tcols * iTile + jTile;
            int kHrowElem = kHrow * (1 + tileBx) + 0 + jTileElem;

            tileHrowMat_gpu[kHrowElem] = j * gapoCost;
        }
    }

    // Initialize score matrix header column.
    {
        // The tile header row and column have an extra zeroth element that needs initializing.
        // That's why we divide by (1 + ...).
        int jTile = 0;
        int iTile = tid / (1 + tileBy);

        if (iTile < trows)
        {
            int iTileElem = tid % (1 + tileBy);
            int i = iTile * tileBy + iTileElem;

            int kHcol = tcols * iTile + jTile; // row-major
            int kHcolElem = kHcol * (1 + tileBy) + 0 + iTileElem;

            tileHcolMat_gpu[kHcolElem] = i * gapoCost;
        }
    }
}

// Calculate the score matrix.
// The score matrix is represented as two matrices (row-major order):
// + tile header row matrix,
// + tile header column matrix.
__global__ static void Nw_Gpu9_KernelB(
    // standard params
    const int* const seqX_gpu,
    const int* const seqY_gpu,
    int* const tileHrowMat_gpu,
    int* const tileHcolMat_gpu,
    const int* const subst_gpu,
    const int substsz,
    const int gapoCost,
    // params related to tile B
    const int trows,
    const int tcols,
    const int tileBx,
    const int tileBy,
    const int subtileCntX,
    const int subtileCntY,
    const int subtileBx,
    const int d)
{
    extern __shared__ int shmem[/* substsz*substsz + tileBx + tileBy + (1+tileBx) + (1+tileBy) */];
    int* const subst /*[substsz*substsz]*/ = shmem + 0;
    int* const seqX /*[tileBx]*/ = subst + substsz * substsz;
    int* const seqY /*[tileBy]*/ = seqX + tileBx;
    int* const tileHrow /*[(1+tileBx)]*/ = seqY + tileBy;
    int* const tileHcol /*[(1+tileBy)]*/ = tileHrow + (1 + tileBx);

    // Initialize the substitution matrix in shared memory.
    {
        int i = threadIdx.x;
        while (i < substsz * substsz)
        {
            el(subst, substsz, 0, i) = el(subst_gpu, substsz, 0, i);
            i += blockDim.x;
        }
    }

    // Block should finish initializing substitution matrix in shared memory.
    __syncthreads();

    // Tile schematic:
    //       x x x x x
    //       | | | | |
    //     h h h h h h
    // y --h . . . . .
    // y --h . . . . .
    // y --h . . . . .
    // Observe that the X and Y seqences don't need to be extended by 1 to the left and by 1 to the top.

    // Map a tile on the current tile diagonal to this thread block.
    // (d,t) -- tile coordinates in the grid of tiles (score matrix).
    int tbeg = max(0, d - (tcols - 1));
    int tend = min(d + 1, trows);
    int t = tbeg + blockIdx.x;

    // Initialize the tile's window into the global X sequence.
    {
        int jbeg = 1 + (d - t) * tileBx;

        int j = threadIdx.x;
        while (j < tileBx)
        {
            seqX[j] = seqX_gpu[jbeg + j];
            j += blockDim.x;
        }
    }

    // Initialize the tile's window into the global Y sequence.
    {
        int ibeg = 1 + (t)*tileBy;

        int i = threadIdx.x;
        while (i < tileBy)
        {
            seqY[i] = seqY_gpu[ibeg + i];
            i += blockDim.x;
        }
    }

    // Initialize the tile's header row in shared memory.
    {
        int iTile = t;
        int jTile = d - t;
        int kHrow = tcols * iTile + jTile;
        int jbeg = kHrow * (1 + tileBx);

        int j = threadIdx.x;
        while (j < 1 + tileBx)
        {
            tileHrow[j] = tileHrowMat_gpu[jbeg + j];
            j += blockDim.x;
        }
    }

    // Initialize the tile's header column in shared memory.
    {
        int iTile = t;
        int jTile = d - t;
        int kHcol = tcols * iTile + jTile; // row-major
        int ibeg = kHcol * (1 + tileBy);

        int i = threadIdx.x;
        while (i < 1 + tileBy)
        {
            tileHcol[i] = tileHcolMat_gpu[ibeg + i];
            i += blockDim.x;
        }
    }

    // Block should finish initializing the tile's windows into the X and Y sequences and its header row and column.
    __syncthreads();

    // Calculate the subtile elements.
    // Subtile shematic:
    //               |h  h  h  h  h  h  h  h  h  |.  .  .  .  .
    //             . |h  .  .  .  .  x  x  o  .  |.  .  .  .
    //          .  . |h  .  .  . |ul u| o  .  .  |.  .  .
    //       .  .  . |h  .  .  x |l  c| .  .  .  |.  .
    //    .  .  .  . |h  .  x  x  o  .  .  .  .  |.
    // .  .  .  .  . |h  .  x  o  .  .  .  .  .  |
    // Observe that we only need three elements from the previous two diagonals, to slide the calculation window to the right each iteration.
    // Therefore each thread keeps them in its registers.
    //
    // Warp thread 0 will read from the subtile header row each iteration, and share that value with the other threads.
    // Warp thread 31 will write its current value to the last subtile row (reusing subtile header row for this purpose).
    // Each warp thread, upon reaching the subtile right boundary, will write its current value to the last subtile column (reusing subtile header column for this purpose).

    // Save the up-left and left element before any calculation to simplify the algorithm.
    // The thread now only needs to worry how it's going to get its 'up' element.
    //
    // Observe that we initialized the elements as if we slid once to the left (in the reverse direction).
    // That's because the code below first slides to the right, and then calculates the current element.
    int upleft = 0;
    int left = 0;
    int up = tileHcol[1 + (threadIdx.x - 1)];
    int curr = tileHcol[1 + (threadIdx.x)];

    int warpIdxX = threadIdx.x % warpSize;
    // The current thread's position in the tile.
    // Append artificial tiles on the left, so that we have a jagged pattern.
    // These tiles will be only used for thread block synchronization, not calculated.
    int i = threadIdx.x;
    int jbeg = 0 - threadIdx.x - (threadIdx.x / warpSize) * subtileBx;
    // All subtile threads have to calculate the same number of elements. Otherwise, warp sync would deadlock.
    // "Artificial" elements outside the tile boundaries are of no concern for correctness.
    int jend = jbeg + subtileCntX * subtileBx;
    int j = jbeg;
    // De-skew the subtile diagonals so it's a rectangle. These variables see the tile as such.
    int jInSubtile = 0;
    int jInSubtileEnd = subtileBx;

    // Zeroth block thread should update the upper-left corner header elements.
    // These will not be updated during the calculation, because none of the threads will map to them.
    // Syncronization is unnecessary, since only the zeroth block thread can access tileHcol[0].
    if (i == 0 && j == 0)
    {
        tileHrow[0] = tileHcol[1 + (tileBy - 1)];
        tileHcol[0] = tileHrow[1 + (tileBx - 1)];
    }

    while (j < jend)
    {
        // Synchronize thread block on the start of every subtile diagonal.
        if (jInSubtile == 0)
        {
            __syncthreads();

            // All bounds are inclusive.
            int jSubtileUpLeft = j + warpIdxX;
            int jSubtileDownLeft = j - (warpSize - 1 - warpIdxX);
            int jSubtileUpRight = jSubtileUpLeft + subtileBx - (1 /*inclusive*/);
            int jSubtileDownRight = jSubtileDownLeft + subtileBx - (1 /*inclusive*/);

            // Skip calculating subtiles made up of only artificial elements.
            if (jSubtileUpRight < 0 || jSubtileDownLeft > tileBx - 1)
            {
                j += subtileBx;
                continue;
            }
            // From this point on we must be in a "real" subtile (contains real elements to be calculated).

            // Correctness: offset the warp diagonal to the right until we touch the first "real" element with the zeroth warp thread.
            // Otherwise we would lose the 'up' element (and the 'up left' element one warp sync later)
            // in the __shfl_up_sync call below before we even start the first "real" calculation.
            if (jSubtileUpLeft < 0)
            {
                int delta = -jSubtileUpLeft;
                j += delta;
                jInSubtile += delta;
            }

            // Optimization - limit the subtile to the right, so as to not unnecessarily calculate completely artificial subtile diagonals.
            if (jSubtileDownRight > tileBx - 1)
            {
                int delta = jSubtileDownRight - (tileBx - 1);
                jInSubtileEnd -= delta;
            }
        }

        // Prevent losing the 'left' header element before the first "real" calculation.
        if (j >= 0 && j < tileBx)
        {
            upleft = up;
            left = curr;
        }

        // Syncs thread memory accesses in the warp.
        // Initialize 'up' elements for all warp threads except the zeroth.
        // + Copies from a lane with lower thread id relative to the caller thread id.
        up = __shfl_up_sync(0xffffffff /*mask*/, curr /*var*/, 1 /*delta*/, warpSize /*width*/);

        // Initialize 'up' element for the zeroth thread.
        // For "artificial" elements, initialize to 0 so that behavior is deterministic.
        if (warpIdxX == 0)
        {
            up = 0;
            if (j >= 0 && j < tileBx)
            {
                up = tileHrow[1 + j];
            }
        }

        if (/*i >= 0 && i < tileBy && */ j >= 0 && j < tileBx)
        {
            curr = upleft + el(subst, substsz, seqY[i], seqX[j]); // MOVE DOWN-RIGHT
            curr = max(curr, up + gapoCost);                      // MOVE DOWN
            curr = max(curr, left + gapoCost);                    // MOVE RIGHT

            if (j == tileBx - 1)
            {
                // When a block thread reaches the right tile boundary, save that element to the next header column.
                tileHcol[1 + i] = curr;
            }

            if (warpIdxX == warpSize - 1)
            {
                // Last subtile thread should save its current element to the next header row.
                tileHrow[1 + j] = curr;
            }
        }

        j++;
        jInSubtile++;
        if (jInSubtile >= jInSubtileEnd)
        {
            jInSubtile = 0;
            jInSubtileEnd = subtileBx;
        }
    }

    // Block should finish calculating this tile.
    __syncthreads();

    // Save the tile last row to the tile header row matrix.
    {
        int iTileBelow = (t) + 1;
        int jTileBelow = (d - t);

        // Bottommost tile should not save its row.
        if (iTileBelow < trows)
        {
            int kHrowBelow = tcols * iTileBelow + jTileBelow;
            int jbeg = kHrowBelow * (1 + tileBx);

            int j = threadIdx.x;
            while (j < 1 + tileBx)
            {
                tileHrowMat_gpu[jbeg + j] = tileHrow[j];
                j += blockDim.x;
            }
        }
    }

    // Save the tile last column to the tile header column matrix.
    {
        int iTileRight = (t);
        int jTileRight = (d - t) + 1;

        // Rightmost tile should not save its column.
        if (jTileRight < tcols)
        {
            int kHcol = tcols * iTileRight + jTileRight; // row-major
            int ibeg = kHcol * (1 + tileBy);

            int i = threadIdx.x;
            while (i < 1 + tileBy)
            {
                tileHcolMat_gpu[ibeg + i] = tileHcol[i];
                i += blockDim.x;
            }
        }
    }
}

// Parallel gpu implementation of the Needleman-Wunsch algorithm.
// The score matrix is represented as two matrices (row-major order):
// + tile header row matrix,
// + tile header column matrix.
//
// Assumes that the row sequence (X) is longer or equal in length to the column sequence (Y).
NwStat NwAlign_Gpu9_Mlsp_DiagDiagDiag(const NwAlgParams& pr, NwAlgInput& nw, NwAlgResult& res)
{
    // Number of threads per block for kernel A.
    int threadsPerBlockA = {};
    // Tile B is a multiple of subtiles B in both dimensions.
    int tileBx = {};
    int tileBy = {};
    int subtileCntX = {};
    int subtileCntY = {};
    // Subtile B must have one dimension be a multiple of the number of threads in a warp.
    int subtileBx = {};
    int subtileBy = nw.warpsz;

    try
    {
        threadsPerBlockA = pr.at("threadsPerBlockA").curr();
        int subtileRows = pr.at("subtileRows").curr();
        int subtileCols = pr.at("subtileCols").curr();
        subtileBx = pr.at("subtileBx").curr();

        // TileB is square, but the subtile is a parallelogram with a 45 degree bottom-left angle.
        tileBx = subtileCols * subtileBx;
        tileBy = subtileRows * subtileBy;
        // Extend tileB width, so that the subtile tiles perfectly the parallelogram 45 degree version of tileB.
        // This maximizes the number of calculations for the same number of whole block synchronizations (equal to subtileCntX).
        int k = (int)ceil(float(tileBx + tileBy - 1) / subtileBx);
        tileBx = k * subtileBx - (tileBy - 1);
        // Now we have a jagged subtiling.
        subtileCntX = k + (subtileRows - 1);
        subtileCntY = subtileRows;

        if (subtileBx < subtileBy || tileBx < tileBy)
        {
            return NwStat::errorInvalidValue;
        }

        if ((threadsPerBlockA < nw.warpsz || threadsPerBlockA > nw.maxThreadsPerBlock) ||
            (subtileRows < 1 || subtileCols < 1 || subtileBx < 1))
        {
            return NwStat::errorInvalidValue;
        }
    }
    catch (const std::out_of_range&)
    {
        return NwStat::errorInvalidValue;
    }

    // Adjusted gpu score matrix dimensions.
    // The matrix dimensions are rounded up to 1 + <the nearest multiple of the tile B size>.
    int adjrows = 1 + tileBy * (int)ceil(float(nw.adjrows - 1) / tileBy);
    int adjcols = 1 + tileBx * (int)ceil(float(nw.adjcols - 1) / tileBx);
    // Special case when very small and very large sequences are compared.
    if (adjrows == 1)
    {
        adjrows = 1 + tileBy;
    }
    if (adjcols == 1)
    {
        adjcols = 1 + tileBx;
    }
    // The number of tiles per row and column of the score matrix.
    int trows = (int)ceil(float(adjrows - 1) / tileBy);
    int tcols = (int)ceil(float(adjcols - 1) / tileBx);

    // Start the timer.
    Stopwatch& sw = res.sw_align;
    sw.start();

    // Allocate space in the ram and gpu global memory.
    try
    {
        nw.seqX_gpu.init(adjcols);
        nw.seqY_gpu.init(adjrows);
        nw.tileHrowMat_gpu.init(trows * tcols * (1 + tileBx));
        nw.tileHcolMat_gpu.init(trows * tcols * (1 + tileBy));

        nw.tileHrowMat.init(trows * tcols * (1 + tileBx));
        nw.tileHcolMat.init(trows * tcols * (1 + tileBy));

        std::vector<int> tmpTile((1 + tileBy) * (1 + tileBx), 0);
        std::swap(nw.tile, tmpTile);
    }
    catch (const std::exception&)
    {
        return NwStat::errorMemoryAllocation;
    }

    // Measure allocation time.
    sw.lap("align.alloc");

    // Copy data from host to device.
    if (hipSuccess != (res.cudaStat = memTransfer(nw.seqX_gpu, nw.seqX, nw.adjcols)))
    {
        return NwStat::errorMemoryTransfer;
    }
    if (hipSuccess != (res.cudaStat = memTransfer(nw.seqY_gpu, nw.seqY, nw.adjrows)))
    {
        return NwStat::errorMemoryTransfer;
    }
    // Also initialize padding, since it is used to access elements in the substitution matrix.
    if (hipSuccess != (res.cudaStat = memSet(nw.seqX_gpu, nw.adjcols, 0 /*value*/)))
    {
        return NwStat::errorMemoryTransfer;
    }
    if (hipSuccess != (res.cudaStat = memSet(nw.seqY_gpu, nw.adjrows, 0 /*value*/)))
    {
        return NwStat::errorMemoryTransfer;
    }

    // Measure memory transfer time.
    sw.lap("align.cpy_dev");

    //  x x x x x x
    //  x . . . . .
    //  x . . . . .
    //  x . . . . .
    // Launch kernel A to initialize the score matrix's header row and column.
    // The score matrix is represented as two matrices (row-major order):
    // + tile header row matrix,
    // + tile header column matrix.
    {
        // Size of shared memory per block in bytes.
        int shmemByteSize = (0);

        dim3 blockDim {};
        blockDim.x = threadsPerBlockA;

        // Calculate the necessary number of blocks to cover the larger score matrix dimension.
        dim3 gridDim {};
        {
            int tileHrowMat_RowElemCount = tcols * (1 + tileBx);
            int tileHcolMat_ColElemCount = trows * (1 + tileBy);
            int largerDimElemCount = max2(tileHrowMat_RowElemCount, tileHcolMat_ColElemCount);
            gridDim.x = (int)ceil(float(largerDimElemCount) / threadsPerBlockA);
        }

        int* tileHrowMat_gpu = nw.tileHrowMat_gpu.data();
        int* tileHcolMat_gpu = nw.tileHcolMat_gpu.data();

        void* kargs[] {
            &tileHrowMat_gpu,
            &tileHcolMat_gpu,
            &trows,
            &tcols,
            &tileBx,
            &tileBy,
            &nw.gapoCost};

        if (hipSuccess != (res.cudaStat = hipLaunchKernel((void*)Nw_Gpu9_KernelA, gridDim, blockDim, kargs, shmemByteSize, hipStreamDefault)))
        {
            return NwStat::errorKernelFailure;
        }
    }

    // Wait for the gpu to finish before going to the next step.
    if (hipSuccess != (res.cudaStat = hipDeviceSynchronize()))
    {
        return NwStat::errorKernelFailure;
    }

    // Measure header initialization time.
    sw.lap("align.init_hdr");

    //  x x x x x x       x x x x x x       x x x x x x
    //  x / / / . .       x . . . / /       x . . . . .|/ /
    //  x / / . . .   +   x . . / / .   +   x . . . . /|/
    //  x / . . . .       x . / / . .       x . . . / /|
    // Launch kernel B for each (minor) tile diagonal of the score matrix.
    {
        hipStream_t stream;
        if (hipSuccess != (res.cudaStat = hipStreamCreate(&stream)))
        {
            return NwStat::errorKernelFailure;
        }
        auto defer1 = make_defer([&stream]() noexcept
        {
            hipStreamDestroy(stream);
        });

        hipGraph_t graph;
        if (hipSuccess != (res.cudaStat = hipGraphCreate(&graph, 0)))
        {
            return NwStat::errorKernelFailure;
        }
        auto defer2 = make_defer([&graph]() noexcept
        {
            hipGraphDestroy(graph);
        });

        // start capturing kernel launches by this thread
        if (hipSuccess != (res.cudaStat = hipStreamBeginCapture(stream, hipStreamCaptureModeThreadLocal)))
        {
            return NwStat::errorKernelFailure;
        }

        // Size of shared memory per block in bytes.
        int shmemsz = (
            /*subst[]*/ nw.substsz * nw.substsz * sizeof(int)
            /*seqX[]*/
            + tileBx * sizeof(int)
            /*seqY[]*/
            + tileBy * sizeof(int)
            /*tileHrow[]*/
            + (1 + tileBx) * sizeof(int)
            /*tileHcol[]*/
            + (1 + tileBy) * sizeof(int));

        dim3 blockB {};
        {
            blockB.x = nw.warpsz * subtileCntY;
        }

        // For all (minor) tile diagonals in the score matrix.
        for (int d = 0; d < tcols - 1 + trows; d++)
        {
            dim3 gridB {};
            {
                int tbeg = max2(0, d - (tcols - 1));
                int tend = min2(d + 1, trows);
                // Number of tiles on the current (minor) tile diagonal.
                int dsize = tend - tbeg;

                gridB.x = dsize;
            }

            int* seqX_gpu = nw.seqX_gpu.data();
            int* seqY_gpu = nw.seqY_gpu.data();
            int* tileHrowMat_gpu = nw.tileHrowMat_gpu.data();
            int* tileHcolMat_gpu = nw.tileHcolMat_gpu.data();
            int* subst_gpu = nw.subst_gpu.data();

            void* kargs[] {
                // standard params
                &seqX_gpu,
                &seqY_gpu,
                &tileHrowMat_gpu,
                &tileHcolMat_gpu,
                &subst_gpu,
                &nw.substsz,
                &nw.gapoCost,
                // params related to tile B
                &trows,
                &tcols,
                &tileBx,
                &tileBy,
                &subtileCntX,
                &subtileCntY,
                &subtileBx,
                //&subtileBy,
                &d};

            if (hipSuccess != (res.cudaStat = hipLaunchKernel((void*)Nw_Gpu9_KernelB, gridB, blockB, kargs, shmemsz, stream)))
            {
                return NwStat::errorKernelFailure;
            }
        }

        // collect kernel launches from this thread
        if (hipSuccess != (res.cudaStat = hipStreamEndCapture(stream, &graph)))
        {
            return NwStat::errorKernelFailure;
        }

        hipGraphExec_t graphExec;
        if (hipSuccess != (res.cudaStat = hipGraphInstantiate(&graphExec, graph, nullptr /*pErrorNode*/, nullptr /*pLogBuffer*/, 0 /*bufferSize*/)))
        {
            return NwStat::errorKernelFailure;
        }
        auto defer3 = make_defer([&graphExec]() noexcept
        {
            hipGraphExecDestroy(graphExec);
        });

        // actually execute the kernels
        if (hipSuccess != (res.cudaStat = hipGraphLaunch(graphExec, hipStreamDefault)))
        {
            return NwStat::errorKernelFailure;
        }
    }

    // Wait for the gpu to finish before going to the next step.
    if (hipSuccess != (res.cudaStat = hipDeviceSynchronize()))
    {
        return NwStat::errorKernelFailure;
    }

    // Measure calculation time.
    sw.lap("align.calc");

    // Save the calculated score matrix.
    nw.tileHdrMatRows = trows;
    nw.tileHdrMatCols = tcols;
    nw.tileHrowLen = 1 + tileBx;
    nw.tileHcolLen = 1 + tileBy;

    if (hipSuccess != (res.cudaStat = memTransfer(nw.tileHrowMat, nw.tileHrowMat_gpu, trows * tcols * (1 + tileBx))))
    {
        return NwStat::errorMemoryTransfer;
    }
    if (hipSuccess != (res.cudaStat = memTransfer(nw.tileHcolMat, nw.tileHcolMat_gpu, trows * tcols * (1 + tileBy))))
    {
        return NwStat::errorMemoryTransfer;
    }

    // Measure memory transfer time.
    sw.lap("align.cpy_host");

    TileAndElemIJ co;
    NwTrace2_GetTileAndElemIJ(nw, nw.adjrows - 1 /*last valid i pos*/, nw.adjcols - 1 /*last valid j pos*/, co);
    NwTrace2_AlignTile(nw.tile, nw, co);
    res.align_cost = el(nw.tile, 1 + tileBx, co.iTileElem, co.jTileElem);

    // Increment calculation time.
    sw.lap("align.calc");

    return NwStat::success;
}
