#include "hip/hip_runtime.h"
#include "defer.hpp"
#include "math.hpp"
#include "nw_fns.hpp"
#include "nwalign_shared.hpp"
#include "run_types.hpp"
#include <cmath>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <vector>

// Initialize the score matrix's header row and column.
// The score matrix is represented as two matrices (row-major order):
// + tile header row matrix,
// + tile header column matrix.
__global__ static void Nw_Gpu8_KernelA(
    int* const tileHrowMat_gpu,
    int* const tileHcolMat_gpu,
    const int trows,
    const int tcols,
    const int tileBx,
    const int tileBy,
    const int gapoCost)
{
    int tid = (blockDim.x * blockIdx.x + threadIdx.x);

    // Initialize score matrix header row.
    {
        // The tile header row and column have an extra zeroth element that needs initializing.
        // That's why we divide by (1 + ...).
        int jTile = tid / (1 + tileBx);
        int iTile = 0;

        if (jTile < tcols)
        {
            int jTileElem = tid % (1 + tileBx);
            int j = jTile * tileBx + jTileElem;

            int kHrow = tcols * iTile + jTile;
            int kHrowElem = kHrow * (1 + tileBx) + 0 + jTileElem;

            tileHrowMat_gpu[kHrowElem] = j * gapoCost;
        }
    }

    // Initialize score matrix header column.
    {
        // The tile header row and column have an extra zeroth element that needs initializing.
        // That's why we divide by (1 + ...).
        int jTile = 0;
        int iTile = tid / (1 + tileBy);

        if (iTile < trows)
        {
            int iTileElem = tid % (1 + tileBy);
            int i = iTile * tileBy + iTileElem;

            int kHcol = tcols * iTile + jTile; // row-major
            int kHcolElem = kHcol * (1 + tileBy) + 0 + iTileElem;

            tileHcolMat_gpu[kHcolElem] = i * gapoCost;
        }
    }
}

// Calculate the score matrix.
// The score matrix is represented as two matrices (row-major order):
// + tile header row matrix,
// + tile header column matrix.
__global__ static void Nw_Gpu8_KernelB(
    // standard params
    const int* const seqX_gpu,
    const int* const seqY_gpu,
    int* const tileHrowMat_gpu,
    int* const tileHcolMat_gpu,
    const int* const subst_gpu,
    const int substsz,
    const int gapoCost,
    // params related to tile B
    const int trows,
    const int tcols,
    const int tileBx,
    const int tileBy,
    const int d)
{
    extern __shared__ int shmem[/* substsz*substsz + tileBx + tileBy + (1+tileBx) + (1+tileBy) */];
    int* const subst /*[substsz*substsz]*/ = shmem + 0;
    int* const seqX /*[tileBx]*/ = subst + substsz * substsz;
    int* const seqY /*[tileBy]*/ = seqX + tileBx;
    int* const tileHrow /*[(1+tileBx)]*/ = seqY + tileBy;
    int* const tileHcol /*[(1+tileBy)]*/ = tileHrow + (1 + tileBx);

    // Initialize the substitution matrix in shared memory.
    {
        int i = threadIdx.x;
        while (i < substsz * substsz)
        {
            el(subst, substsz, 0, i) = el(subst_gpu, substsz, 0, i);
            i += blockDim.x;
        }
    }

    // Block should finish initializing substitution matrix in shared memory.
    __syncthreads();

    // Tile schematic:
    //       x x x x x
    //       | | | | |
    //     h h h h h h
    // y --h . . . . .
    // y --h . . . . .
    // y --h . . . . .
    // Observe that the X and Y seqences don't need to be extended by 1 to the left and by 1 to the top.

    // Map a tile on the current tile diagonal to this thread block.
    // (d,t) -- tile coordinates in the grid of tiles (score matrix).
    int tbeg = max(0, d - (tcols - 1));
    int tend = min(d + 1, trows);
    int t = tbeg + blockIdx.x;

    // Initialize the tile's window into the global X sequence.
    {
        int jbeg = 1 + (d - t) * tileBx;

        int j = threadIdx.x;
        while (j < tileBx)
        {
            seqX[j] = seqX_gpu[jbeg + j];
            j += blockDim.x;
        }
    }

    // Initialize the tile's window into the global Y sequence.
    {
        int ibeg = 1 + (t)*tileBy;

        int i = threadIdx.x;
        while (i < tileBy)
        {
            seqY[i] = seqY_gpu[ibeg + i];
            i += blockDim.x;
        }
    }

    // Initialize the tile's header row in shared memory.
    {
        int iTile = t;
        int jTile = d - t;
        int kHrow = tcols * iTile + jTile;
        int jbeg = kHrow * (1 + tileBx);

        int j = threadIdx.x;
        while (j < 1 + tileBx)
        {
            tileHrow[j] = tileHrowMat_gpu[jbeg + j];
            j += blockDim.x;
        }
    }

    // Initialize the tile's header column in shared memory.
    {
        int iTile = t;
        int jTile = d - t;
        int kHcol = tcols * iTile + jTile; // row-major
        int ibeg = kHcol * (1 + tileBy);

        int i = threadIdx.x;
        while (i < 1 + tileBy)
        {
            tileHcol[i] = tileHcolMat_gpu[ibeg + i];
            i += blockDim.x;
        }
    }

    // Block should finish initializing the tile's windows into the X and Y sequences and its header row and column.
    __syncthreads();

    // Calculate the tile elements.
    // Only threads in the first warp from this block are active here, other warps have to wait.
    if (threadIdx.x < warpSize)
    {
        // Tile shematic:
        //               |h  h  h  h  h  h  h  h  h  |.  .  .  .  .
        //             . |h  .  .  .  .  x  x  o  .  |.  .  .  .
        //          .  . |h  .  .  . |ul u| o  .  .  |.  .  .
        //       .  .  . |h  .  .  x |l  c| .  .  .  |.  .
        //    .  .  .  . |h  .  x  x  o  .  .  .  .  |.
        // .  .  .  .  . |h  .  x  o  .  .  .  .  .  |
        // Observe that we only need three elements from the previous two diagonals, to slide the calculation window to the right each iteration.
        // Therefore each thread keeps them in its registers.
        //
        // Warp thread 0 will read from the tile header row each iteration, and share that value with the other threads.
        // Warp thread 31 will write its current value to the last tile row (reusing tile header row for this purpose).
        // Each warp thread, upon reaching the tile right boundary, will write its current value to the last tile column (reusing tile header column for this purpose).

        // Save the up-left and left element before any calculation to simplify the algorithm.
        // The thread now only needs to worry how it's going to get its 'up' element.
        //
        // Observe that we initialized the elements as if we slid once to the left (in the reverse direction).
        // That's because the code below first slides to the right, and then calculates the current element.
        int upleft = 0;
        int left = 0;
        int up = tileHcol[1 + (threadIdx.x - 1)];
        int curr = tileHcol[1 + (threadIdx.x)];

        // The current thread's position in the tile.
        const int i = threadIdx.x;
        int j = 0 - threadIdx.x;
        // All threads have to calculate the same number of elements. Otherwise, warp sync would deadlock.
        // "Artificial" elements outside the tile boundaries are of no concern for correctness.
        int jend = j + (tileBx + (tileBy - 1));

        // Zeroth warp thread should update the upper-left corner header elements.
        // These will not be updated during the calculation, because none of the threads will map to them.
        if (i == 0 && j == 0)
        {
            tileHrow[0] = tileHcol[1 + (tileBy - 1)];
            tileHcol[0] = tileHrow[1 + (tileBx - 1)];
        }

        while (j < jend)
        {
            // Prevent losing the 'left' header element before the first "real" calculation.
            if (j >= 0 && j < tileBx)
            {
                upleft = up;
                left = curr;
            }

            // Initialize 'up' elements for all warp threads except the zeroth.
            // (Copies from a lane with lower thread id relative to the caller thread id.
            // Also syncs the threads in the warp.)
            up = __shfl_up_sync(0xffffffff /*mask*/, curr /*var*/, 1 /*delta*/, warpSize /*width*/);

            // Initialize 'up' element for the zeroth thread.
            // For "artificial" elements, initialize to 0 so that behavior is deterministic.
            if (i == 0)
            {
                up = (j >= 0 && j < tileBx) ? tileHrow[1 + j] : 0;
            }

            if (/*i >= 0 && i < tileBy && */ j >= 0 && j < tileBx)
            {
                curr = upleft + el(subst, substsz, seqY[i], seqX[j]); // MOVE DOWN-RIGHT
                curr = max(curr, up + gapoCost);                      // MOVE DOWN
                curr = max(curr, left + gapoCost);                    // MOVE RIGHT

                if (j == tileBx - 1)
                {
                    // When a warp thread reaches the right tile boundary, save that element to the next header column.
                    tileHcol[1 + i] = curr;
                }

                if (i == tileBy - 1)
                {
                    // Last warp thread should save its current element to the next header row.
                    tileHrow[1 + j] = curr;
                }
            }

            j++;
        }
    }

    // Block should finish calculating this tile.
    __syncthreads();

    // Save the tile last row to the tile header row matrix.
    {
        int iTileBelow = (t) + 1;
        int jTileBelow = (d - t);

        // Bottommost tile should not save its row.
        if (iTileBelow < trows)
        {
            int kHrowBelow = tcols * iTileBelow + jTileBelow;
            int jbeg = kHrowBelow * (1 + tileBx);

            int j = threadIdx.x;
            while (j < 1 + tileBx)
            {
                tileHrowMat_gpu[jbeg + j] = tileHrow[j];
                j += blockDim.x;
            }
        }
    }

    // Save the tile last column to the tile header column matrix.
    {
        int iTileRight = (t);
        int jTileRight = (d - t) + 1;

        // Rightmost tile should not save its column.
        if (jTileRight < tcols)
        {
            int kHcol = tcols * iTileRight + jTileRight; // row-major
            int ibeg = kHcol * (1 + tileBy);

            int i = threadIdx.x;
            while (i < 1 + tileBy)
            {
                tileHcolMat_gpu[ibeg + i] = tileHcol[i];
                i += blockDim.x;
            }
        }
    }
}

// Parallel gpu implementation of the Needleman-Wunsch algorithm.
// The score matrix is represented as two matrices (row-major order):
// + tile header row matrix,
// + tile header column matrix.
//
// Assumes that the row sequence (X) is longer or equal in length to the column sequence (Y).
NwStat NwAlign_Gpu8_Mlsp_DiagDiag(const NwAlgParams& pr, NwAlgInput& nw, NwAlgResult& res)
{
    // Number of threads per block for kernel A.
    int threadsPerBlockA {};
    // Tile B must have one dimension fixed to the number of threads in a warp.
    int tileBx {};
    int tileBy {nw.warpsz};
    // Reduce the number of warps in the thread block in kernel B.
    int warpDivFactorB {};

    // Get parameters.
    try
    {
        threadsPerBlockA = pr.at("threadsPerBlockA").curr();
        tileBx = pr.at("tileBx").curr();
        warpDivFactorB = pr.at("warpDivFactorB").curr();

        if ((threadsPerBlockA < nw.warpsz || threadsPerBlockA > nw.maxThreadsPerBlock) ||
            (tileBx < 1 || warpDivFactorB < 1))
        {
            return NwStat::errorInvalidValue;
        }
    }
    catch (const std::exception&)
    {
        return NwStat::errorInvalidValue;
    }

    // Adjusted gpu score matrix dimensions.
    // The matrix dimensions are rounded up to 1 + <the nearest multiple of the tile B size>.
    int adjrows = 1 + tileBy * (int)ceil(float(nw.adjrows - 1) / tileBy);
    int adjcols = 1 + tileBx * (int)ceil(float(nw.adjcols - 1) / tileBx);
    // Special case when very small and very large sequences are compared.
    if (adjrows == 1)
    {
        adjrows = 1 + tileBy;
    }
    if (adjcols == 1)
    {
        adjcols = 1 + tileBx;
    }
    // The number of tiles per row and column of the score matrix.
    int trows = (int)ceil(float(adjrows - 1) / tileBy);
    int tcols = (int)ceil(float(adjcols - 1) / tileBx);

    // Start the timer.
    Stopwatch& sw = res.sw_align;
    sw.start();

    // Allocate.
    try
    {
        nw.seqX_gpu.init(adjcols);
        nw.seqY_gpu.init(adjrows);
        nw.tileHrowMat_gpu.init(trows * tcols * (1 + tileBx));
        nw.tileHcolMat_gpu.init(trows * tcols * (1 + tileBy));

        nw.tileHrowMat.init(trows * tcols * (1 + tileBx));
        nw.tileHcolMat.init(trows * tcols * (1 + tileBy));

        std::vector<int> tmpTile((1 + tileBy) * (1 + tileBx), 0);
        std::swap(nw.tile, tmpTile);
    }
    catch (const std::exception&)
    {
        return NwStat::errorMemoryAllocation;
    }

    updateNwAlgPeakMemUsage(nw, res);

    // Measure allocation time.
    sw.lap("align.alloc");

    // Copy data from host to device.
    if (hipSuccess != (res.cudaStat = memTransfer(nw.seqX_gpu, nw.seqX, nw.adjcols)))
    {
        return NwStat::errorMemoryTransfer;
    }
    if (hipSuccess != (res.cudaStat = memTransfer(nw.seqY_gpu, nw.seqY, nw.adjrows)))
    {
        return NwStat::errorMemoryTransfer;
    }
    // Also initialize padding, since it is used to access elements in the substitution matrix.
    if (hipSuccess != (res.cudaStat = memSet(nw.seqX_gpu, nw.adjcols, 0 /*value*/)))
    {
        return NwStat::errorMemoryTransfer;
    }
    if (hipSuccess != (res.cudaStat = memSet(nw.seqY_gpu, nw.adjrows, 0 /*value*/)))
    {
        return NwStat::errorMemoryTransfer;
    }

    // Measure memory transfer time.
    sw.lap("align.cpy_dev");

    //  x x x x x x
    //  x . . . . .
    //  x . . . . .
    //  x . . . . .
    // Launch kernel A to initialize the score matrix's header row and column.
    // The score matrix is represented as two matrices (row-major order):
    // + tile header row matrix,
    // + tile header column matrix.
    {
        // Size of shared memory per block in bytes.
        dim3 blockA {};
        dim3 gridA {};
        size_t shmemsz {};

        blockA.x = threadsPerBlockA;

        hipFuncAttributes attr {};
        if (hipSuccess != (res.cudaStat = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((void*))Nw_Gpu8_KernelA)))
        {
            return NwStat::errorKernelFailure;
        }

        int maxActiveBlocksPerSm = 0;
        if (hipSuccess != (res.cudaStat = hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocksPerSm, (void*)Nw_Gpu8_KernelA, blockA.x, shmemsz)))
        {
            return NwStat::errorKernelFailure;
        }

        // Calculate the necessary number of blocks to cover the larger score matrix dimension.
        {
            int tileHrowMat_RowElemCount = tcols * (1 + tileBx);
            int tileHcolMat_ColElemCount = trows * (1 + tileBy);
            int largerDimElemCount = max2(tileHrowMat_RowElemCount, tileHcolMat_ColElemCount);
            gridA.x = (int)ceil(float(largerDimElemCount) / threadsPerBlockA);
        }

        int maxActiveBlocksActual = min2(maxActiveBlocksPerSm * nw.sm_count, (int)gridA.x);
        updateNwAlgPeakMemUsage(nw, res, &attr, maxActiveBlocksActual, blockA.x, shmemsz);

        int* tileHrowMat_gpu = nw.tileHrowMat_gpu.data();
        int* tileHcolMat_gpu = nw.tileHcolMat_gpu.data();

        void* kargs[] {
            &tileHrowMat_gpu,
            &tileHcolMat_gpu,
            &trows,
            &tcols,
            &tileBx,
            &tileBy,
            &nw.gapoCost};

        if (hipSuccess != (res.cudaStat = hipLaunchKernel((void*)Nw_Gpu8_KernelA, gridA, blockA, kargs, shmemsz, hipStreamDefault)))
        {
            return NwStat::errorKernelFailure;
        }
    }

    // Wait for the gpu to finish before going to the next step.
    if (hipSuccess != (res.cudaStat = hipDeviceSynchronize()))
    {
        return NwStat::errorKernelFailure;
    }

    // Measure header initialization time.
    sw.lap("align.init_hdr");

    //  x x x x x x       x x x x x x       x x x x x x
    //  x / / / . .       x . . . / /       x . . . . .|/ /
    //  x / / . . .   +   x . . / / .   +   x . . . . /|/
    //  x / . . . .       x . / / . .       x . . . / /|
    // Launch kernel B for each (minor) tile diagonal of the score matrix.
    {
        hipStream_t stream {};
        if (hipSuccess != (res.cudaStat = hipStreamCreate(&stream)))
        {
            return NwStat::errorKernelFailure;
        }
        auto defer1 = make_defer([&stream]() noexcept
        {
            hipStreamDestroy(stream);
        });

        hipGraph_t graph {};
        if (hipSuccess != (res.cudaStat = hipGraphCreate(&graph, 0)))
        {
            return NwStat::errorKernelFailure;
        }
        auto defer2 = make_defer([&graph]() noexcept
        {
            hipGraphDestroy(graph);
        });

        // start capturing kernel launches by this thread
        if (hipSuccess != (res.cudaStat = hipStreamBeginCapture(stream, hipStreamCaptureModeThreadLocal)))
        {
            return NwStat::errorKernelFailure;
        }
        hipError_t cudaStreamEndCapture_stat = hipSuccess;
        auto defer3_cudaStreamEndCapture = make_defer([&cudaStreamEndCapture_stat, &stream, &graph]() noexcept
        {
            cudaStreamEndCapture_stat = hipStreamEndCapture(stream, &graph);
        });

        dim3 blockB {};
        dim3 gridB {};

        {
            // The number of threads should be divisible by the warp size.
            // But for performance reasons, we don't need all those single-use warps, just half of them (or some other fraction).
            // That way the thread block can be smaller while doing the same amount of work.
            int warps = (int)ceil(float(max2(tileBx, tileBy)) / nw.warpsz / warpDivFactorB);
            blockB.x = nw.warpsz * warps;
        }

        // Size of shared memory per block in bytes.
        size_t shmemsz =
            /*subst[]*/ nw.substsz * nw.substsz * sizeof(int)
            /*seqX[]*/
            + tileBx * sizeof(int)
            /*seqY[]*/
            + tileBy * sizeof(int)
            /*tileHrow[]*/
            + (1 + tileBx) * sizeof(int)
            /*tileHcol[]*/
            + (1 + tileBy) * sizeof(int);

        hipFuncAttributes attr {};
        if (hipSuccess != (res.cudaStat = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((void*))Nw_Gpu8_KernelB)))
        {
            return NwStat::errorKernelFailure;
        }

        int maxActiveBlocksPerSm = 0;
        if (hipSuccess != (res.cudaStat = hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocksPerSm, (void*)Nw_Gpu8_KernelB, blockB.x, shmemsz)))
        {
            return NwStat::errorKernelFailure;
        }

        // For all (minor) tile diagonals in the score matrix.
        for (int d = 0; d < tcols - 1 + trows; d++)
        {
            {
                int tbeg = max2(0, d - (tcols - 1));
                int tend = min2(d + 1, trows);
                // Number of tiles on the current (minor) tile diagonal.
                int dsize = tend - tbeg;

                gridB.x = dsize;
            }

            int maxActiveBlocksActual = min2(maxActiveBlocksPerSm * nw.sm_count, (int)gridB.x);
            updateNwAlgPeakMemUsage(nw, res, &attr, maxActiveBlocksActual, blockB.x, shmemsz);

            int* seqX_gpu = nw.seqX_gpu.data();
            int* seqY_gpu = nw.seqY_gpu.data();
            int* tileHrowMat_gpu = nw.tileHrowMat_gpu.data();
            int* tileHcolMat_gpu = nw.tileHcolMat_gpu.data();
            int* subst_gpu = nw.subst_gpu.data();

            void* kargs[] {
                // standard params
                &seqX_gpu,
                &seqY_gpu,
                &tileHrowMat_gpu,
                &tileHcolMat_gpu,
                &subst_gpu,
                &nw.substsz,
                &nw.gapoCost,
                // params related to tile B
                &trows,
                &tcols,
                &tileBx,
                &tileBy,
                &d};

            if (hipSuccess != (res.cudaStat = hipLaunchKernel((void*)Nw_Gpu8_KernelB, gridB, blockB, kargs, shmemsz, stream)))
            {
                return NwStat::errorKernelFailure;
            }
        }

        // collect kernel launches from this thread
        defer3_cudaStreamEndCapture();
        if (hipSuccess != (res.cudaStat = cudaStreamEndCapture_stat))
        {
            return NwStat::errorKernelFailure;
        }

        hipGraphExec_t graphExec;
        if (hipSuccess != (res.cudaStat = hipGraphInstantiate(&graphExec, graph, nullptr /*pErrorNode*/, nullptr /*pLogBuffer*/, 0 /*bufferSize*/)))
        {
            return NwStat::errorKernelFailure;
        }
        auto defer4 = make_defer([&graphExec]() noexcept
        {
            hipGraphExecDestroy(graphExec);
        });

        // actually execute the kernels
        if (hipSuccess != (res.cudaStat = hipGraphLaunch(graphExec, hipStreamDefault)))
        {
            return NwStat::errorKernelFailure;
        }
    }

    // Wait for the gpu to finish before going to the next step.
    if (hipSuccess != (res.cudaStat = hipDeviceSynchronize()))
    {
        return NwStat::errorKernelFailure;
    }

    // Measure calculation time.
    sw.lap("align.calc");

    // Save the calculated score matrix.
    nw.tileHdrMatRows = trows;
    nw.tileHdrMatCols = tcols;
    nw.tileHrowLen = 1 + tileBx;
    nw.tileHcolLen = 1 + tileBy;

    if (hipSuccess != (res.cudaStat = memTransfer(nw.tileHrowMat, nw.tileHrowMat_gpu, trows * tcols * (1 + tileBx))))
    {
        return NwStat::errorMemoryTransfer;
    }
    if (hipSuccess != (res.cudaStat = memTransfer(nw.tileHcolMat, nw.tileHcolMat_gpu, trows * tcols * (1 + tileBy))))
    {
        return NwStat::errorMemoryTransfer;
    }

    // Measure memory transfer time.
    sw.lap("align.cpy_host");

    TileAndElemIJ co;
    NwTrace2_GetTileAndElemIJ(nw, nw.adjrows - 1 /*last valid i pos*/, nw.adjcols - 1 /*last valid j pos*/, co);
    NwTrace2_AlignTile(nw.tile, nw, co);
    res.align_cost = el(nw.tile, 1 + tileBx, co.iTileElem, co.jTileElem);

    // Increment calculation time.
    sw.lap("align.calc");

    return NwStat::success;
}
