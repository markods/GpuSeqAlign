#include "hip/hip_runtime.h"
#include "defer.hpp"
#include "fmt_guard.hpp"
#include "math.hpp"
#include "nwalign_shared.hpp"
#include "run_types.hpp"
#include <hip/hip_runtime.h>

// cuda kernel for the parallel implementation
__global__ static void Nw_Gpu1_KernelAB(
    const int* const seqX_gpu,
    const int* const seqY_gpu,
    int* const score_gpu,
    const int* const subst_gpu,
    const int adjrows,
    const int adjcols,
    const int substsz,
    const int gapoCost,
    const int d // the current minor diagonal in the score matrix (exclude the header row and column)
)
{
    // the dimensions of the matrix without its row and column header
    const int rows = -1 + adjrows;
    const int cols = -1 + adjcols;

    //  x x x x x x       x x x x x x       x x x x x x
    //  x / / / . .       x . . . / /       x . . . . .|/ /
    //  x / / . . .   +   x . . / / .   +   x . . . . /|/
    //  x / . . . .       x . / / . .       x . . . / /|

    // (d,p) -- element coordinates on the score matrix diagonal
    int pbeg = max(0, d - (cols - 1));
    int pend = min(d + 1, rows);
    // position of the current thread's element on the matrix diagonal
    int p = pbeg + (blockDim.x * blockIdx.x + threadIdx.x);

    // if the thread maps onto an element on the current matrix diagonal
    if (p < pend)
    {
        // position of the current element
        int i = 1 + (p);
        int j = 1 + (d - p);

        // if the thread maps onto the start of the diagonal
        if (d < cols && p == 0)
        {
            // initialize TOP header element
            el(score_gpu, adjcols, 0, j) = j * gapoCost;
            // if this is also the zeroth diagonal (with only one element on it)
            if (d == 0)
            {
                // initialize TOP-LEFT header element
                el(score_gpu, adjcols, 0, 0) = 0 * gapoCost;
            }
        }
        // if the thread maps onto the end of the diagonal
        if (d < rows && p == pend - 1)
        {
            // initialize LEFT header element
            el(score_gpu, adjcols, i, 0) = i * gapoCost;
        }

        // calculate the current element's value
        // +   always subtract the insert delete cost from the result, since that value was added to the initial temporary
        int p0 = el(subst_gpu, substsz, seqY_gpu[i], seqX_gpu[j]) - gapoCost;

        int p1 = el(score_gpu, adjcols, i - 1, j - 1) + p0; // MOVE DOWN-RIGHT
        int p2 = max(el(score_gpu, adjcols, i - 1, j), p1); // MOVE DOWN
        int p3 = max(el(score_gpu, adjcols, i, j - 1), p2); // MOVE RIGHT
        el(score_gpu, adjcols, i, j) = p3 + gapoCost;
    }
}

// parallel gpu implementation of the Needleman-Wunsch algorithm
NwStat NwAlign_Gpu1_Ml_Diag(const NwAlgParams& pr, NwAlgInput& nw, NwAlgResult& res)
{
    // number of threads per block
    // +   the tile is one-dimensional
    int threadsPerBlock {};

    // Get parameters.
    try
    {
        threadsPerBlock = pr.at("threadsPerBlock").curr();
        if (threadsPerBlock < nw.warpsz || threadsPerBlock > nw.maxThreadsPerBlock)
        {
            return NwStat::errorInvalidValue;
        }
    }
    catch (const std::exception&)
    {
        return NwStat::errorInvalidValue;
    }

    // adjusted gpu score matrix dimensions
    // +   the matrix dimensions are rounded up to 1 + the nearest multiple of the tile A size (in order to be evenly divisible)
    int adjrows = nw.adjrows;
    int adjcols = nw.adjcols;
    // special case when very small and very large sequences are compared
    if (adjrows == 1)
    {
        adjrows = 2;
    }
    if (adjcols == 1)
    {
        adjcols = 2;
    }
    // the dimensions of the matrix without its row and column header
    int rows = -1 + adjrows;
    int cols = -1 + adjcols;

    // start the timer
    Stopwatch& sw = res.sw_align;
    sw.start();

    // Allocate.
    try
    {
        nw.seqX_gpu.init(adjcols);
        nw.seqY_gpu.init(adjrows);
        nw.score_gpu.init(adjrows * adjcols);

        nw.score.init(nw.adjrows * nw.adjcols);
    }
    catch (const std::exception&)
    {
        return NwStat::errorMemoryAllocation;
    }

    updateNwAlgPeakMemUsage(nw, res);

    // measure allocation time
    sw.lap("align.alloc");

    // copy data from host to device
    if (hipSuccess != (res.cudaStat = memTransfer(nw.seqX_gpu, nw.seqX, nw.adjcols)))
    {
        return NwStat::errorMemoryTransfer;
    }
    if (hipSuccess != (res.cudaStat = memTransfer(nw.seqY_gpu, nw.seqY, nw.adjrows)))
    {
        return NwStat::errorMemoryTransfer;
    }
    // also initialize padding, since it is used to access elements in the substitution matrix
    if (hipSuccess != (res.cudaStat = memSet(nw.seqX_gpu, nw.adjcols, 0 /*value*/)))
    {
        return NwStat::errorMemoryTransfer;
    }
    if (hipSuccess != (res.cudaStat = memSet(nw.seqY_gpu, nw.adjrows, 0 /*value*/)))
    {
        return NwStat::errorMemoryTransfer;
    }

    // measure memory transfer time
    sw.lap("align.cpy_dev");

    //  x x x x x x       x x x x x x       x x x x x x
    //  x / / / . .       x . . . / /       x . . . . .|/ /
    //  x / / . . .   +   x . . / / .   +   x . . . . /|/
    //  x / . . . .       x . / / . .       x . . . / /|
    // launch kernel for each minor diagonal of the score matrix
    {
        hipStream_t stream {};
        if (hipSuccess != (res.cudaStat = hipStreamCreate(&stream)))
        {
            return NwStat::errorKernelFailure;
        }
        auto defer1 = make_defer([&stream]() noexcept
        {
            hipStreamDestroy(stream);
        });

        hipGraph_t graph {};
        if (hipSuccess != (res.cudaStat = hipGraphCreate(&graph, 0)))
        {
            return NwStat::errorKernelFailure;
        }
        auto defer2 = make_defer([&graph]() noexcept
        {
            hipGraphDestroy(graph);
        });

        // start capturing kernel launches by this thread
        if (hipSuccess != (res.cudaStat = hipStreamBeginCapture(stream, hipStreamCaptureModeThreadLocal)))
        {
            return NwStat::errorKernelFailure;
        }
        hipError_t cudaStreamEndCapture_stat = hipSuccess;
        auto defer3_cudaStreamEndCapture = make_defer([&cudaStreamEndCapture_stat, &stream, &graph]() noexcept
        {
            cudaStreamEndCapture_stat = hipStreamEndCapture(stream, &graph);
        });

        // grid and block dimensions for kernel
        dim3 gridA {};
        dim3 blockA {};
        // calculate size of shared memory per block in bytes
        size_t shmemsz {};

        // take the number of threads per block as the only dimension
        blockA.x = threadsPerBlock;

        hipFuncAttributes attr {};
        if (hipSuccess != (res.cudaStat = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((void*))Nw_Gpu1_KernelAB)))
        {
            return NwStat::errorKernelFailure;
        }

        int maxActiveBlocksPerSm = 0;
        if (hipSuccess != (res.cudaStat = hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocksPerSm, (void*)Nw_Gpu1_KernelAB, blockA.x, shmemsz)))
        {
            return NwStat::errorKernelFailure;
        }

        // for all minor diagonals in the score matrix (excluding the header row and column)
        for (int d = 0; d < cols - 1 + rows; d++)
        {
            // calculate grid and block dimensions for kernel
            {
                int pbeg = max2(0, d - (cols - 1));
                int pend = min2(d + 1, rows);

                // the number of elements on the current diagonal
                int dsize = pend - pbeg;

                // take the number of blocks on the current score matrix diagonal as the only dimension
                // +   launch at least one block on the x axis
                gridA.x = (int)ceil(float(dsize) / threadsPerBlock);
            }

            int maxActiveBlocksActual = min2(maxActiveBlocksPerSm * nw.sm_count, (int)gridA.x);
            updateNwAlgPeakMemUsage(nw, res, &attr, maxActiveBlocksActual, blockA.x, shmemsz);

            // create variables for gpu arrays in order to be able to take their addresses
            int* seqX_gpu = nw.seqX_gpu.data();
            int* seqY_gpu = nw.seqY_gpu.data();
            int* score_gpu = nw.score_gpu.data();
            int* subst_gpu = nw.subst_gpu.data();

            // group arguments to be passed to kernel
            void* kargs[] {
                &seqX_gpu,
                &seqY_gpu,
                &score_gpu,
                &subst_gpu,
                &adjrows,
                &adjcols,
                &nw.substsz,
                &nw.gapoCost,
                &d};

            // launch the kernel in the given stream (don't statically allocate shared memory)
            if (hipSuccess != (res.cudaStat = hipLaunchKernel((void*)Nw_Gpu1_KernelAB, gridA, blockA, kargs, shmemsz, stream)))
            {
                return NwStat::errorKernelFailure;
            }
        }

        // collect kernel launches from this thread
        defer3_cudaStreamEndCapture();
        if (hipSuccess != (res.cudaStat = cudaStreamEndCapture_stat))
        {
            return NwStat::errorKernelFailure;
        }

        hipGraphExec_t graphExec;
        if (hipSuccess != (res.cudaStat = hipGraphInstantiate(&graphExec, graph, nullptr /*pErrorNode*/, nullptr /*pLogBuffer*/, 0 /*bufferSize*/)))
        {
            return NwStat::errorKernelFailure;
        }
        auto defer4 = make_defer([&graphExec]() noexcept
        {
            hipGraphExecDestroy(graphExec);
        });

        // actually execute the kernels
        if (hipSuccess != (res.cudaStat = hipGraphLaunch(graphExec, hipStreamDefault)))
        {
            return NwStat::errorKernelFailure;
        }
    }

    // wait for the gpu to finish before going to the next step
    if (hipSuccess != (res.cudaStat = hipDeviceSynchronize()))
    {
        return NwStat::errorKernelFailure;
    }

    // measure calculation time
    sw.lap("align.calc");

    // save the calculated score matrix
    if (hipSuccess != (res.cudaStat = memTransfer(nw.score, nw.score_gpu, nw.adjrows, nw.adjcols, adjcols)))
    {
        return NwStat::errorMemoryTransfer;
    }

    res.align_cost = el(nw.score, nw.adjcols, nw.adjrows - 1, nw.adjcols - 1);

    // measure memory transfer time
    sw.lap("align.cpy_host");

    return NwStat::success;
}
