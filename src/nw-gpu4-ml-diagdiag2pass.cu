#include "hip/hip_runtime.h"
#include "common.hpp"

// cuda kernel A for the parallel implementation
// +   initializes the score matrix in the gpu
__global__ static void Nw_Gpu4_KernelA(
    const int *const seqX_gpu,
    const int *const seqY_gpu,
    int *const score_gpu,
    const int *const subst_gpu,
    const int adjrows,
    const int adjcols,
    const int substsz,
    const int indel,
    const unsigned tileAx,
    const unsigned tileAy)
{
    extern __shared__ int shmem[/* substsz*substsz + tileAx + tileAy */];
    // the substitution matrix and relevant parts of the two sequences
    // +   stored in shared memory for faster random access
    // NOTE: should we align allocations to 0-th shared memory bank?
    int *const subst /*[substsz*substsz]*/ = shmem + 0;
    int *const seqX /*[tileAx]*/ = subst + substsz * substsz;
    int *const seqY /*[tileAy]*/ = seqX + tileAx;

    // start position of the block in the global X and Y sequences
    int ibeg = blockIdx.y * tileAy;
    int jbeg = blockIdx.x * tileAx;
    // real tile size (since the score matrix is not evenly divisible by tileA, but is instead by tileB)
    int realAy = min(tileAy, adjrows - ibeg);
    int realAx = min(tileAx, adjcols - jbeg);

    // initialize the substitution shared memory copy
    {
        // map the threads from the thread block onto the substitution matrix elements
        int i = threadIdx.x;
        // while the current thread maps onto an element in the matrix
        while (i < substsz * substsz)
        {
            // copy the current element from the global substitution matrix
            el(subst, substsz, 0, i) = el(subst_gpu, substsz, 0, i);
            // map this thread to the next element with stride equal to the number of threads in this block
            i += blockDim.x;
        }
    }

    // initialize the X and Y sequences' shared memory copies
    {
        // map the threads from the thread block onto the global X sequence's elements (which will be used in this tile)
        int j = threadIdx.x;
        // while the current thread maps onto an element in the tile's X sequence
        for (; j < realAx; j += blockDim.x)
        {
            // initialize that element in the X seqence's shared window
            seqX[j] = seqX_gpu[jbeg + j];
        }
        // while the current thread maps onto the padding of the tile's X sequence
        for (; j < tileAx; j += blockDim.x)
        {
            // initialize that element in the X seqence's padding
            seqX[j] = 0;
        }

        // map the threads from the thread block onto the global Y sequence's elements (which will be used in this tile)
        int i = threadIdx.x;
        // while the current thread maps onto an element in the tile's Y sequence
        for (; i < realAy; i += blockDim.x)
        {
            // initialize that element in the Y seqence's shared window
            seqY[i] = seqY_gpu[ibeg + i];
        }
        // while the current thread maps onto the padding of the tile's Y sequence
        for (; i < tileAy; i += blockDim.x)
        {
            // initialize that element in the Y seqence's padding
            seqY[i] = seqY_gpu[ibeg + i];
        }
    }

    // make sure that all threads have finished initializing their corresponding elements
    __syncthreads();

    // initialize the score matrix in global memory
    {
        // the current element value
        int elem = 0;

        // while this thread maps onto an element in the current tile in the global score matrix
        // +   blockDim.x is the number of threads in the block (in other words stride)
        int i = threadIdx.x / realAx;
        int di = blockDim.x / realAx;
        int j = threadIdx.x % realAx;
        int dj = blockDim.x % realAx;
        while (i < realAy)
        {
            // the position of the element this thread currently maps to in the score matrix
            int ipos = ibeg + i;
            int jpos = jbeg + j;
            // if the current thread is not in the first row or column of the score matrix
            // +   use the substitution matrix to calculate the score matrix element value
            // +   increase the value by insert delete cost, since then the formula for calculating the actual element value in kernel B becomes simpler
            if (ipos > 0 && jpos > 0)
            {
                elem = el(subst, substsz, seqY[i], seqX[j]) - indel;
            }
            // otherwise, if the current thread is in the first row or column
            // +   update the score matrix element using the insert delete cost
            else
            {
                elem = (ipos | jpos) * indel;
            }

            // update the corresponding element in global memory
            // +   fully coallesced memory access
            el(score_gpu, adjcols, ipos, jpos) = elem;

            // map the current thread to the next tile element
            i += di;
            j += dj;
            // if the column index is out of bounds, increase the row index by one and wrap around the column index
            if (j >= realAx)
            {
                i++;
                j -= realAx;
            }
        }
    }
}

// cuda kernel B for the parallel implementation
// +   calculates the score matrix in the gpu using the initialized score matrix from kernel A
// +   the given matrix minus the padding (zeroth row and column) must be evenly divisible by the tile B
__global__ static void Nw_Gpu4_KernelB(
    int *const score_gpu,
    const int indel,
    const int warpsz,
    const int trows,
    const int tcols,
    const unsigned tileBx,
    const unsigned tileBy,
    const int d // the current minor tile diagonal in the score matrix (exclude the header row and column)
)
{
    extern __shared__ int shmem[/* (1+tileBy)*(1+tileBx) */];
    // matrix tile which this thread block maps onto
    // +   stored in shared memory for faster random access
    int *const tile /*[(1+tileBy)*(1+tileBx)]*/ = shmem + 0;

    //  / / / . .       . . . / /       . . . . .|/ /
    //  / / . . .   +   . . / / .   +   . . . . /|/
    //  / . . . .       . / / . .       . . . / /|

    // map a tile on the current tile diagonal to this thread block
    {
        // (s,t) -- tile coordinates in the grid of tiles (score matrix)
        int tbeg = max(0, d - (tcols - 1));
        int tend = min(d, trows - 1);

        // map a tile on the current diagonal of tiles to this thread block
        // +   then go to the next tile on the diagonal with stride equal to the number of thread blocks in the thread grid
        int t = tbeg + blockIdx.x;
        // initialize the score matrix tile
        {
            // position of the top left element of the current tile in the score matrix
            int ibeg = (1 + (t)*tileBy) - 1;
            int jbeg = (1 + (d - t) * tileBx) - 1;
            // the number of columns in the score matrix
            int adjcols = 1 + tcols * tileBx;

            // current thread position in the tile
            int i = threadIdx.x / (tileBx + 1);
            int j = threadIdx.x % (tileBx + 1);
            // stride on the current thread position in the tile, equal to the number of threads in this thread block
            // +   it is split into row and column increments for the thread's position for performance reasons (avoids using division and modulo operator in the inner cycle)
            int di = blockDim.x / (tileBx + 1);
            int dj = blockDim.x % (tileBx + 1);

            // while the current thread maps onto an element in the tile
            while (i < (1 + tileBy))
            {
                // copy the current element from the global score matrix to the tile
                el(tile, 1 + tileBx, i, j) = el(score_gpu, adjcols, ibeg + i, jbeg + j);

                // map the current thread to the next tile element
                i += di;
                j += dj;
                // if the column index is out of bounds, increase the row index by one and wrap around the column index
                if (j >= (1 + tileBx))
                {
                    i++;
                    j -= (1 + tileBx);
                }
            }
        }

        // all threads in this block should finish initializing this tile in shared memory
        __syncthreads();

        // calculate the tile elements
        // +   only threads in the first warp from this block are active here, other warps have to wait
        if (threadIdx.x < warpsz)
        {
            // the number of rows and columns in the tile without its first row and column (the part of the tile to be calculated)
            int rows = tileBy;
            int cols = tileBx;

            //  x x x x x x       x x x x x x       x x x x x x
            //  x / / / . .       x . . . / /       x . . . . .|/ /
            //  x / / . . .   +   x . . / / .   +   x . . . . /|/
            //  x / . . . .       x . / / . .       x . . . / /|

            // for all diagonals in the tile without its first row and column
            for (int d = 0; d < cols - 1 + rows; d++)
            {
                // (d,p) -- element coordinates in the tile
                int pbeg = max(0, d - (cols - 1));
                int pend = min(d, rows - 1);
                // position of the current thread's element on the tile diagonal
                int p = pbeg + threadIdx.x;

                // if the thread maps onto an element on the current tile diagonal
                if (p <= pend)
                {
                    // position of the current element
                    int i = 1 + (p);
                    int j = 1 + (d - p);

                    // calculate the current element's value
                    // +   always subtract the insert delete cost from the result, since the kernel A added that value to each element of the score matrix
                    int temp1 = el(tile, 1 + tileBx, i - 1, j - 1) + el(tile, 1 + tileBx, i, j);
                    int temp2 = max(el(tile, 1 + tileBx, i - 1, j), el(tile, 1 + tileBx, i, j - 1));
                    el(tile, 1 + tileBx, i, j) = max(temp1, temp2) + indel;
                }

                // all threads in this warp should finish calculating the tile's current diagonal
                __syncwarp();
            }
        }

        // all threads in this block should finish calculating this tile
        __syncthreads();

        // save the score matrix tile
        {
            // position of the first (top left) calculated element of the current tile in the score matrix
            int ibeg = (1 + (t)*tileBy);
            int jbeg = (1 + (d - t) * tileBx);
            // the number of columns in the score matrix
            int adjcols = 1 + tcols * tileBx;

            // current thread position in the tile
            int i = threadIdx.x / tileBx;
            int j = threadIdx.x % tileBx;
            // stride on the current thread position in the tile, equal to the number of threads in this thread block
            // +   it is split into row and column increments for the thread's position for performance reasons (avoids using division and modulo operator in the inner cycle)
            int di = blockDim.x / tileBx;
            int dj = blockDim.x % tileBx;

            // while the current thread maps onto an element in the tile
            while (i < tileBy)
            {
                // copy the current element from the tile to the global score matrix
                el(score_gpu, adjcols, ibeg + i, jbeg + j) = el(tile, 1 + tileBx, 1 + i, 1 + j);

                // map the current thread to the next tile element
                i += di;
                j += dj;
                // if the column index is out of bounds, increase the row index by one and wrap around the column index
                if (j >= tileBx)
                {
                    i++;
                    j -= tileBx;
                }
            }
        }

        // all threads in this block should finish saving this tile
        __syncthreads();
    }
}

// parallel gpu implementation of the Needleman-Wunsch algorithm
NwStat NwAlign_Gpu4_Ml_DiagDiag2Pass(NwParams &pr, NwInput &nw, NwResult &res)
{
    // tile sizes for kernels A and B
    // +   tile A should have one dimension be a multiple of the warp size for full memory coallescing
    // +   tile B must have one dimension fixed to the number of threads in a warp
    unsigned tileAx;
    unsigned tileAy;
    unsigned tileBx;
    unsigned tileBy;

    // get the parameter values
    try
    {
        tileAx = pr["tileAx"].curr();
        tileAy = pr["tileAy"].curr();
        tileBx = pr["tileBx"].curr();
        tileBy = pr["tileBy"].curr();
    }
    catch (const std::out_of_range &ex)
    {
        return NwStat::errorInvalidValue;
    }

    if (tileBx != nw.warpsz && tileBy != nw.warpsz)
    {
        return NwStat::errorInvalidValue;
    }

    // adjusted gpu score matrix dimensions
    // +   the matrix dimensions are rounded up to 1 + the nearest multiple of the tile B size (in order to be evenly divisible)
    int adjrows = 1 + tileBy * ceil(float(nw.adjrows - 1) / tileBy);
    int adjcols = 1 + tileBx * ceil(float(nw.adjcols - 1) / tileBx);
    // special case when very small and very large sequences are compared
    if (adjrows == 1)
    {
        adjrows = 1 + tileBy;
    }
    if (adjcols == 1)
    {
        adjcols = 1 + tileBx;
    }

    // start the timer
    Stopwatch &sw = res.sw_align;
    sw.start();

    // reserve space in the ram and gpu global memory
    try
    {
        nw.seqX_gpu.init(adjcols);
        nw.seqY_gpu.init(adjrows);
        nw.score_gpu.init(adjrows * adjcols);

        nw.score.init(nw.adjrows * nw.adjcols);
    }
    catch (const std::exception &ex)
    {
        return NwStat::errorMemoryAllocation;
    }

    // measure allocation time
    sw.lap("alloc");

    // copy data from host to device
    if (hipSuccess != (cudaStatus = memTransfer(nw.seqX_gpu, nw.seqX, nw.adjcols)))
    {
        return NwStat::errorMemoryTransfer;
    }
    if (hipSuccess != (cudaStatus = memTransfer(nw.seqY_gpu, nw.seqY, nw.adjrows)))
    {
        return NwStat::errorMemoryTransfer;
    }
    // also initialize padding, since it is used to access elements in the substitution matrix
    if (hipSuccess != (cudaStatus = memSet(nw.seqX_gpu, nw.adjcols, 0 /*value*/)))
    {
        return NwStat::errorMemoryTransfer;
    }
    if (hipSuccess != (cudaStatus = memSet(nw.seqY_gpu, nw.adjrows, 0 /*value*/)))
    {
        return NwStat::errorMemoryTransfer;
    }

    // measure memory transfer time
    sw.lap("cpy-dev");

    // launch kernel A
    {
        // calculate grid dimensions for kernel A
        dim3 gridA{};
        gridA.y = ceil(float(adjrows) / tileAy);
        gridA.x = ceil(float(adjcols) / tileAx);
        // block dimensions for kernel A
        unsigned threadsPerBlockA = min(nw.maxThreadsPerBlock, tileAy * tileAx);
        dim3 blockA{threadsPerBlockA};

        // calculate size of shared memory per block in bytes
        int shmemsz = (
            /*subst[][]*/ nw.substsz * nw.substsz * sizeof(int)
            /*seqX[]*/
            + tileAx * sizeof(int)
            /*seqY[]*/
            + tileAy * sizeof(int));

        // create variables for gpu arrays in order to be able to take their addresses
        int *seqX_gpu = nw.seqX_gpu.data();
        int *seqY_gpu = nw.seqY_gpu.data();
        int *score_gpu = nw.score_gpu.data();
        int *subst_gpu = nw.subst_gpu.data();

        // group arguments to be passed to kernel A
        void *kargs[]{
            &seqX_gpu,
            &seqY_gpu,
            &score_gpu,
            &subst_gpu,
            &adjrows,
            &adjcols,
            &nw.substsz,
            &nw.indel,
            &tileAx,
            &tileAy};

        // launch the kernel in the given stream (don't statically allocate shared memory)
        if (hipSuccess != (cudaStatus = hipLaunchKernel((void *)Nw_Gpu4_KernelA, gridA, blockA, kargs, shmemsz, nullptr /*stream*/)))
        {
            return NwStat::errorKernelFailure;
        }
    }

    // wait for the gpu to finish before going to the next step
    if (hipSuccess != (cudaStatus = hipDeviceSynchronize()))
    {
        return NwStat::errorKernelFailure;
    }

    // measure calculation init time
    sw.lap("calc-1");

    //  x x x x x x       x x x x x x       x x x x x x
    //  x / / / . .       x . . . / /       x . . . . .|/ /
    //  x / / . . .   +   x . . / / .   +   x . . . . /|/
    //  x / . . . .       x . / / . .       x . . . / /|
    // launch kernel B for each minor tile diagonal of the score matrix
    {
        // grid and block dimensions for kernel B
        dim3 gridB{};
        dim3 blockB{};
        // the number of tiles per row and column of the score matrix
        int trows = ceil(float(adjrows - 1) / tileBy);
        int tcols = ceil(float(adjcols - 1) / tileBx);

        // calculate size of shared memory per block in bytes
        int shmemsz = (
            /*tile[]*/ (1 + tileBy) * (1 + tileBx) * sizeof(int));

        // for all minor tile diagonals in the score matrix (excluding the header row and column)
        for (int d = 0; d < tcols - 1 + trows; d++)
        {
            // calculate grid and block dimensions for kernel B
            {
                int pbeg = max(0, d - (tcols - 1));
                int pend = min(d, trows - 1);

                // the number of elements on the current diagonal
                int dsize = pend - pbeg + 1;

                // take the number of threads on the largest diagonal of the tile
                // +   multiply by the number of half warps in the larger dimension for faster writing to global gpu memory
                blockB.x = nw.warpsz * ceil(max(tileBy, tileBx) * 2. / nw.warpsz);

                // take the number of blocks on the current score matrix diagonal as the only dimension
                // +   launch at least one block on the x axis
                gridB.x = dsize;
            }

            // create variables for gpu arrays in order to be able to take their addresses
            int *seqX_gpu = nw.seqX_gpu.data();
            int *seqY_gpu = nw.seqY_gpu.data();
            int *score_gpu = nw.score_gpu.data();
            int *subst_gpu = nw.subst_gpu.data();

            // group arguments to be passed to kernel B
            void* kargs[]{
                &score_gpu,
                &nw.indel,
                &nw.warpsz,
                &trows,
                &tcols,
                &tileBx,
                &tileBy,
                &d };

            // launch the kernel B in the given stream (don't statically allocate shared memory)
            if (hipSuccess != (cudaStatus = hipLaunchKernel((void *)Nw_Gpu4_KernelB, gridB, blockB, kargs, shmemsz, nullptr /*stream*/)))
            {
                return NwStat::errorKernelFailure;
            }
        }
    }

    // wait for the gpu to finish before going to the next step
    if (hipSuccess != (cudaStatus = hipDeviceSynchronize()))
    {
        return NwStat::errorKernelFailure;
    }

    // measure calculation time
    sw.lap("calc-2");

    // save the calculated score matrix
    if (hipSuccess != (cudaStatus = memTransfer(nw.score, nw.score_gpu, nw.adjrows, nw.adjcols, adjcols)))
    {
        return NwStat::errorMemoryTransfer;
    }

    // measure memory transfer time
    sw.lap("cpy-host");

    return NwStat::success;
}
