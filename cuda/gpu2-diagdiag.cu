#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include "common.h"

// stream used through the rest of the program
#define STREAM_ID 0
// number of streaming multiprocessors (sm-s) and cores per sm
#define MPROCS 28
#define CORES 128
// number of threads in warp
#define WARPSZ 32
// tile sizes for kernels A and B
// +   tile A should have one dimension be a multiple of the warp size for full memory coallescing
// +   tile B must have one dimension fixed to the number of threads in a warp
const int tileAx = 1*WARPSZ;
const int tileAy = 32;
const int tileBx = 60;
const int tileBy = WARPSZ;


// cuda kernel A for the parallel implementation
// +   initializes the score matrix in the gpu
__global__ static void kernelA( int* seqX_gpu, int* seqY_gpu, int* score_gpu, int rows, int cols, int (*blosum62_gpu)[BLOSUMSZ], int insdelcost )
{
   // the blosum matrix and relevant parts of the two sequences
   // +   stored in shared memory for faster random access
   __shared__ int blosum62[BLOSUMSZ][BLOSUMSZ];
   __shared__ int seqX[tileAx];
   __shared__ int seqY[tileAy];

   // initialize the blosum shared memory copy
   {
      // map the threads from the thread block onto the blosum matrix elements
      int i = threadIdx.y*BLOSUMSZ + threadIdx.x;
      // while the current thread maps onto an element in the matrix
      while( i < BLOSUMSZ*BLOSUMSZ )
      {
         // copy the current element from the global blosum matrix
         blosum62[ 0 ][ i ] = blosum62_gpu[ 0 ][ i ];
         // map this thread to the next element with stride equal to the number of threads in this block
         i += tileAy*tileAx;
      }
   }

   // initialize the X and Y sequences' shared memory copies
   {
      // position of the current thread in the global X and Y sequences
      int x = blockIdx.x*blockDim.x;
      int y = blockIdx.y*blockDim.y;
      // map the threads from the first            row  to the shared X sequence part
      // map the threads from the second and later rows to the shared Y sequence part
      int iX = ( threadIdx.y     )*tileAx + threadIdx.x;
      int iY = ( threadIdx.y - 1 )*tileAx + threadIdx.x;

      // if the current thread maps to the first row, initialize the corresponding element
      if( iX < tileAx )        seqX[ iX ] = seqX_gpu[ x + iX ];
      // otherwise, remap it to the first column and initialize the corresponding element
      else if( iY < tileAy )   seqY[ iY ] = seqY_gpu[ y + iY ];
   }
   
   // make sure that all threads have finished initializing their corresponding elements
   __syncthreads();

   // initialize the score matrix in global memory
   {
      // position of the current thread in the score matrix
      int i = blockIdx.y*blockDim.y + threadIdx.y;
      int j = blockIdx.x*blockDim.x + threadIdx.x;
      // position of the current thread in the sequences
      int iX = threadIdx.x;
      int iY = threadIdx.y;
      // the current element value
      int elem = 0;
      
      // if the current thread is outside the score matrix, return
      if( i >= rows || j >= cols ) return;

      // if the current thread is not in the first row or column of the score matrix
      // +   use the blosum matrix to calculate the score matrix element value
      // +   increase the value by insert delete cost, since then the formula for calculating the actual element value in kernel B becomes simpler
      if( i > 0 && j > 0 ) { elem = blosum62[ seqY[iY] ][ seqX[iX] ] + insdelcost; }
      // otherwise, if the current thread is in the first row or column
      // +   update the score matrix element using the insert delete cost
      else                 { elem = -( i|j )*insdelcost; }
      
      // update the corresponding element in global memory
      // +   fully coallesced memory access
      el(score_gpu,cols, i,j) = elem;
   }
}


// cuda kernel B for the parallel implementation
// +   calculates the score matrix in the gpu using the initialized score matrix from kernel A
// +   the given matrix minus the padding (zeroth row and column) must be evenly divisible by the tile B
__global__ static void kernelB( int* score_gpu, int trows, int tcols, int insdelcost )
{
   // matrix tile which this thread block maps onto
   // +   stored in shared memory for faster random access
   __shared__ int tile[1+tileBy][1+tileBx];

   
   //    |/ / / . .   +   . . . / /   +   . . . . .|/ /
   //   /|/ / . . .   +   . . / / .   +   . . . . /|/
   // / /|/ . . . .   +   . / / . .   +   . . . / /|

   // for all diagonals of tiles in the grid of tiles (score matrix)
   for( int s = 0;   s < tcols-1 + trows;   s++ )
   {
      // (s,t) -- tile coordinates in the grid of tiles (score matrix)
      int tbeg = max( 0, s - (tcols-1) );
      int tend = min( s, trows-1 );


      // map a tile on the current diagonal of tiles to this thread block
      // +   then go to the next tile on the diagonal with stride equal to the number of thread blocks in the thread grid
      for( int t = tbeg + blockIdx.x;   t <= tend;   t += gridDim.x )
      {
         // initialize the score matrix tile
         {
            // position of the top left element of the current tile in the score matrix
            int ibeg = ( 1 + (   t )*tileBy ) - 1;
            int jbeg = ( 1 + ( s-t )*tileBx ) - 1;
            // the number of colums in the score matrix
            int cols = 1 + tcols*tileBx;

            // current thread position in the tile
            int i = threadIdx.x / ( tileBx+1 );
            int j = threadIdx.x % ( tileBx+1 );
            // stride on the current thread position in the tile, equal to the number of threads in this thread block
            // +   it is split into row and column increments for the tread's position for performance reasons (avoids using division and modulo operator in the inner cycle)
            int di = blockDim.x / ( tileBx+1 );
            int dj = blockDim.x % ( tileBx+1 );
            
            // while the current thread maps onto an element in the tile
            while( i < ( 1+tileBy ) )
            {
               // copy the current element from the global score matrix to the tile
               tile[ i ][ j ] = el(score_gpu,cols, ibeg+i,jbeg+j);

               // map the current thread to the next tile element
               i += di; j += dj;
               // if the column index is out of bounds, increase the row index by one and wrap around the column index
               if( j >= ( 1+tileBx ) ) { i++; j -= ( 1+tileBx ); }
            }
         }

         // all threads in this block should finish initializing this tile in shared memory
         __syncthreads();
         
         // calculate the tile elements
         // +   only threads in the first warp from this block are active here, other warps have to wait
         if( threadIdx.x < WARPSZ )
         {
            // the number of rows and colums in the tile without its first row and column (the part of the tile to be calculated)
            int rows = tileBy;
            int cols = tileBx;

            //    |/ / . . .   +   . . / / /   +   . . . . .|/ /
            //   /|/ . . . .   +   . / / / .   +   . . . . /|/
            // / /|. . . . .   +   / / / . .   +   . . . / /|

            // for all diagonals in the tile without its first row and column
            for( int d = 0;   d < cols-1 + rows;   d++ )
            {
               // (d,p) -- element coordinates in the tile
               int tbeg = max( 0, d - (cols-1) );
               int tend = min( d, rows-1 );
               // position of the current thread's element on the tile diagonal
               int p = tbeg + threadIdx.x;

               // if the thread maps onto an element on the current tile diagonal
               if( p <= tend )
               {
                  // position of the current element
                  int i = 1 + (   p );
                  int j = 1 + ( d-p );
                  
                  // calculate the current element's value
                  // +   always subtract the insert delete cost from the result, since the kernel A added that value to each element of the score matrix
                  int temp1  =      tile[i-1][j-1] + tile[i  ][j  ];
                  int temp2  = max( tile[i-1][j  ] , tile[i  ][j-1] );
                  tile[i][j] = max( temp1, temp2 ) - insdelcost;
               }

               // all threads in this warp should finish calculating the tile's current diagonal
               __syncwarp();
            }
         }
         
         // all threads in this block should finish calculating this tile
         __syncthreads();
         

         // save the score matrix tile
         {
            // position of the first (top left) calculated element of the current tile in the score matrix
            int ibeg = ( 1 + (   t )*tileBy );
            int jbeg = ( 1 + ( s-t )*tileBx );
            // the number of colums in the score matrix
            int cols = 1 + tcols*tileBx;

            // current thread position in the tile
            int i = threadIdx.x / tileBx;
            int j = threadIdx.x % tileBx;
            // stride on the current thread position in the tile, equal to the number of threads in this thread block
            // +   it is split into row and column increments for the tread's position for performance reasons (avoids using division and modulo operator in the inner cycle)
            int di = blockDim.x / tileBx;
            int dj = blockDim.x % tileBx;
            
            // while the current thread maps onto an element in the tile
            while( i < tileBy )
            {
               // copy the current element from the tile to the global score matrix
               el(score_gpu,cols, ibeg+i,jbeg+j) = tile[ 1+i ][ 1+j ];

               // map the current thread to the next tile element
               i += di; j += dj;
               // if the column index is out of bounds, increase the row index by one and wrap around the column index
               if( j >= tileBx ) { i++; j -= tileBx; }
            }
         }
         
         // all threads in this block should finish saving this tile
         // +   block synchronization unnecessary since the tiles on the current diagonal are independent
      }

      // all threads in this grid should finish calculating the diagonal of tiles
      cooperative_groups::this_grid().sync();
   }
}


// parallel gpu implementation of the Needleman Wunsch algorithm
void Gpu2_DiagDiag( NWArgs& nw, NWResult& res )
{
   // blosum matrix, sequences which will be compared and the score matrix stored in gpu global memory
   int *blosum62_gpu, *seqX_gpu, *seqY_gpu, *score_gpu;
   // allocate space in the gpu global memory
   hipMalloc( &seqX_gpu,     nw.adjcols            * sizeof( int ) );
   hipMalloc( &seqY_gpu,     nw.adjrows            * sizeof( int ) );
   hipMalloc( &score_gpu,    nw.adjrows*nw.adjcols * sizeof( int ) );
   hipMalloc( &blosum62_gpu, BLOSUMSZ*BLOSUMSZ     * sizeof( int ) );
   // create events for measuring kernel execution time
   hipEvent_t start, stop;
   hipEventCreate( &start );
   hipEventCreate( &stop );

   // start the host timer and initialize the gpu timer
   res.sw.lap( "cpu-start" );
   res.Tgpu = 0;

   // copy data from host to device
	hipMemcpy( seqX_gpu,     nw.seqX,     nw.adjcols     * sizeof( int ), hipMemcpyHostToDevice );
	hipMemcpy( seqY_gpu,     nw.seqY,     nw.adjrows     * sizeof( int ), hipMemcpyHostToDevice );
	hipMemcpy( blosum62_gpu, blosum62, BLOSUMSZ*BLOSUMSZ * sizeof( int ), hipMemcpyHostToDevice );


   // printf("   - processing score matrix in a blocky diagonal fashion\n");


   // launch kernel A
   {
      // calculate grid dimensions for kernel A
      dim3 gridA;
      gridA.y = ceil( float( nw.adjrows )/tileAy );
      gridA.x = ceil( float( nw.adjcols )/tileAx );
      // block dimensions for kernel A
      dim3 blockA { tileAx, tileAy };
      
      // launch the kernel in the given stream (don't statically allocate shared memory)
      // +   capture events around kernel launch as well
      // +   update the stop event when the kernel finishes
      hipEventRecord( start, STREAM_ID );
      kernelA<<< gridA, blockA, 0, STREAM_ID >>>( seqX_gpu, seqY_gpu, score_gpu, nw.adjrows, nw.adjcols, ( int (*)[BLOSUMSZ] )blosum62_gpu, nw.insdelcost );
      hipEventRecord( stop, STREAM_ID );
      hipEventSynchronize( stop );
      
      // kernel A execution time
      float ktimeA;
      // calculate the time between the given events
      hipEventElapsedTime( &ktimeA, start, stop );
      // update the total kernel execution time
      res.Tgpu += ktimeA / 1000./*ms*/;
   }
   
   // wait for the gpu to finish before going to the next step
   hipDeviceSynchronize();


   // launch kernel B
   {
      // grid and block dimensions for kernel B
      dim3 gridB;
      dim3 blockB;
      // the number of tiles per row and column of the score matrix
      int trows = ceil( float( nw.adjrows-1 )/tileBy );
      int tcols = ceil( float( nw.adjcols-1 )/tileBx );
      
      // calculate grid and block dimensions for kernel B
      {
         // take the number of warps on the largest tile diagonal times the number of threads in a warp as the number of threads
         // +   also multiply by the number of half warps in the larger dimension for faster writing to global gpu memory
         blockB.x  = ceil( min( tileBy, tileBx )*1./WARPSZ )*WARPSZ;
         blockB.x *= ceil( max( tileBy, tileBx )*2./WARPSZ );
         // take the number of tiles on the largest score matrix diagonal as the only dimension
         gridB.x = min( trows, tcols );

         // the maximum number of parallel blocks on a streaming multiprocessor
         int maxBlocksPerSm = 0;
         // number of threads per block that the kernel will be launched with
         int numThreads = blockB.x;
         // size of shared memory per block in bytes
         int sharedMemSz = ( ( 1+tileBy )*( 1+tileBx ) )*sizeof( int );

         // calculate the max number of parallel blocks per streaming multiprocessor
         hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxBlocksPerSm, kernelB, numThreads, sharedMemSz );
         // the number of cooperative blocks launched must not exceed the maximum possible number of parallel blocks on the device
         gridB.x = min( gridB.x, MPROCS*maxBlocksPerSm );
      }

      // group arguments to be passed to kernel B
      void* kargs[] { &score_gpu, &trows, &tcols, &nw.insdelcost };
      
      // launch the kernel in the given stream (don't statically allocate shared memory)
      // +   capture events around kernel launch as well
      // +   update the stop event when the kernel finishes
      hipEventRecord( start, STREAM_ID );
      hipLaunchCooperativeKernel( ( void* )kernelB, gridB, blockB, kargs, 0, STREAM_ID );
      hipEventRecord( stop, STREAM_ID );
      hipEventSynchronize( stop );
      
      // kernel B execution time
      float ktimeB;
      // calculate the time between the given events
      hipEventElapsedTime( &ktimeB, start, stop );
      // update the total kernel execution time
      res.Tgpu += ktimeB / 1000./*ms*/;
   }

   // wait for the gpu to finish before going to the next step
   hipDeviceSynchronize();
   // save the calculated score matrix
   // +   waits for the device to finish, then copies data from device to host
   hipMemcpy( nw.score, score_gpu, nw.adjrows*nw.adjcols * sizeof( int ), hipMemcpyDeviceToHost );

   // stop the cpu timer
   res.sw.lap( "cpu-end" );
   res.Tcpu = res.sw.dt( "cpu-end", "cpu-start" );

   
   // free allocated space in the gpu global memory
   hipFree( seqX_gpu );
   hipFree( seqY_gpu );
   hipFree( score_gpu );
   hipFree( blosum62_gpu );
   // free events' memory
   hipEventDestroy( start );
   hipEventDestroy( stop );
}





